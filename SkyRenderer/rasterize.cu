#include "hip/hip_runtime.h"
#include "rasterize.cuh"
#include "Dense"
#include "thrust/extrema.h"
#include "Sampling.h"
#include "Shader.cuh"
#include ""

//�������Ĳ�ֵ������1-u-v,u,v
__host__ __device__ Eigen::Vector3f barycentric(Eigen::Vector2f A, Eigen::Vector2f B, Eigen::Vector2f C, Eigen::Vector2f P)
{
	Eigen::Vector3f u = Eigen::Vector3f(B.x() - A.x(), C.x() - A.x(), A.x() - P.x()).cross(Eigen::Vector3f(B.y() - A.y(), C.y() - A.y(), A.y() - P.y()));// u v 1
	return Eigen::Vector3f(1.f - (u.x() + u.y()) / u.z(), u.x() / u.z(), u.y() / u.z());
}

__global__ void CaculatePixel(FrameBuffer frameBuffer, Varyings* fragDatas, DataTruck* dataTruck, int shaderID, Varyings* vertA, Varyings* vertB, Varyings* vertC, int x0, int y0, int w, int h)
{
	const int x = x0 + blockIdx.x * blockDim.x + threadIdx.x;
	const int y = y0 + blockIdx.y * blockDim.y + threadIdx.y;

	auto a = ComputeScreenPos(&frameBuffer, vertA->positionCS);
	auto b = ComputeScreenPos(&frameBuffer, vertB->positionCS);
	auto c = ComputeScreenPos(&frameBuffer, vertC->positionCS);


	Eigen::Vector3f u = barycentric(Eigen::Vector2f(a.x(), a.y()), Eigen::Vector2f(b.x(), b.y()), Eigen::Vector2f(c.x(), c.y()), Eigen::Vector2f(x, y));

	if (u.x() >= 0 && u.y() >= 0 && u.z() >= 0)
	{
		float depth = u.x() * a.z() + u.y() * b.z() + u.z() * c.z();

		if (shaderID == SKYBOX_SHADER)
		{
			depth = 1.f;
		}

		if (!ZTestAutomic(&frameBuffer, depth, x, y))
		{
			return;
		}

		float alpha = u.x() / vertA->positionCS.w();
		float beta = u.y() / vertB->positionCS.w();
		float gamma = u.z() / vertC->positionCS.w();
		float zn = 1 / (alpha + beta + gamma);

		//��ֵ
		Varyings tmpdata;
		tmpdata.positionWS = zn * (alpha * vertA->positionWS + beta * vertB->positionWS + gamma * vertC->positionWS);
		tmpdata.positionCS = zn * (alpha * vertA->positionCS + beta * vertB->positionCS + gamma * vertC->positionCS);
		tmpdata.normalWS = zn * (alpha * vertA->normalWS + beta * vertB->normalWS + gamma * vertC->normalWS);
		tmpdata.tangentWS = zn * (alpha * vertA->tangentWS + beta * vertB->tangentWS + gamma * vertC->tangentWS);
		tmpdata.binormalWS = zn * (alpha * vertA->binormalWS + beta * vertB->binormalWS + gamma * vertC->binormalWS);
		tmpdata.uv = zn * (alpha * vertA->uv + beta * vertB->uv + gamma * vertC->uv);

		Eigen::Vector4f finalColor = Eigen::Vector4f(0, 0, 0, 0);

		switch (shaderID)
		{
		case NONE:
			break;
		case LAMBERT_SHADER:
			finalColor = LambertFrag(tmpdata, dataTruck, &frameBuffer);
			break;
		case SHADOWMAP_SHADER:
			finalColor = ShadowMapFrag(tmpdata, dataTruck, &frameBuffer);
			break;
		case PBR_SHADER:
			finalColor = PBRFrag(tmpdata, dataTruck, &frameBuffer);
			break;
		case SKYBOX_SHADER:
			finalColor = SkyBoxFrag(tmpdata, dataTruck, &frameBuffer);
			break;
		};

		if (depth == GetZ(&frameBuffer, x, y))
		{
			DrawPoint(&frameBuffer, x, y, finalColor);
		}
	}
}

__global__ void CaculateTrangle(FrameBuffer frameBuffer, Varyings* fragDatas, DataTruck* dataTruck, int shaderID, int trangleNum, int offset)
{
	int idx = offset + (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= trangleNum)
	{
		return;
	}

	auto vertA = fragDatas[idx * 3];
	auto vertB = fragDatas[idx * 3 + 1];
	auto vertC = fragDatas[idx * 3 + 2];

	auto a = ComputeScreenPos(&frameBuffer, vertA.positionCS);
	auto b = ComputeScreenPos(&frameBuffer, vertB.positionCS);
	auto c = ComputeScreenPos(&frameBuffer, vertC.positionCS);
	// caculate AABB box
	int minx = thrust::max(0, thrust::min((int)frameBuffer.m_Width, (int)thrust::min(a.x(), thrust::min(b.x(), c.x()))));
	int miny = thrust::max(0, thrust::min((int)frameBuffer.m_Height, (int)thrust::min(a.y(), thrust::min(b.y(), c.y()))));
	int maxx = thrust::min((int)frameBuffer.m_Width, thrust::max(0, (int)thrust::max(a.x(), thrust::max(b.x(), c.x()))));
	int maxy = thrust::min((int)frameBuffer.m_Height, thrust::max(0, (int)thrust::max(a.y(), thrust::max(b.y(), c.y()))));
	//AABB��Χ�еĿ��
	int w = maxx - minx + 1;
	int h = maxy - miny + 1;

	const int threadNum = 8;

	dim3 blockNum(w / threadNum + (w % threadNum != 0), h / threadNum + (h % threadNum != 0));
	dim3 blockSize(threadNum, threadNum);

	CaculatePixel << <blockNum, blockSize >> > (frameBuffer, fragDatas, dataTruck, shaderID, fragDatas + idx * 3, fragDatas + idx * 3 + 1, fragDatas + idx * 3 + 2, minx, miny, w, h);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("CaculatePixel Failed! : %s\n", hipGetErrorString(cudaStatus));
	}
}
/*
* ������ɫ
* �����豸���ڴ�&���к˺���&д������&�ͷ��豸���ڴ�
*/
hipError_t VertKernel(std::vector<Attributes>* vertDatas, std::vector<Varyings>* fragDatas, DataTruck* dataTruck, int shaderID)
{
	hipError_t cudaStatus;

	Attributes* cudaVertDatas = nullptr;
	Varyings* cudaFragDatas = nullptr;
	DataTruck* cudaDataTruck = nullptr;
	Texture* cudatmptextures = nullptr;

	const int threadNum = 192;
	int vertNum = vertDatas->size();
	int blockNum = vertDatas->size() / threadNum + (vertNum % threadNum != 0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//�������ݵ�GPU�ڴ�
	{
		//��������
		cudaStatus = hipMalloc((void**)&cudaVertDatas, sizeof(Attributes) * vertDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaVertDatas");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudaVertDatas, vertDatas->data(), vertNum * sizeof(Attributes), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed! cudaVertDatas");
			goto Error;
		}

		//������ɫ������
		cudaStatus = hipMalloc((void**)&cudaFragDatas, sizeof(Varyings) * fragDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaFragDatas");
			goto Error;
		}
		//dataTruck
		/*
		* �ṹ���к�ָ����������ڴ淽���ο�����
		* https://devforum.nvidia.cn/forum.php?mod=viewthread&tid=6820&extra=&page=1
		*/

		cudaStatus = hipMalloc((void**)&cudatmptextures, dataTruck->texNum * sizeof(Texture));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudatmptextures");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudatmptextures, dataTruck->textures, dataTruck->texNum * sizeof(Texture), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "cudaMemecpy failed! cudatmptextures");
			goto Error;
		}

		auto tmpDataTruck = *dataTruck;
		tmpDataTruck.textures = cudatmptextures;

		cudaStatus = hipMalloc((void**)&cudaDataTruck, sizeof(DataTruck));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaDataTruck");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudaDataTruck, &tmpDataTruck, sizeof(DataTruck), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed! cudaDataTruck");
			goto Error;
		}
	}

	//����Kernel����
	switch (shaderID)
	{
	case NONE:
		break;
	case LAMBERT_SHADER:
		LambertVert << <blockNum, threadNum >> > (cudaVertDatas, cudaFragDatas, cudaDataTruck, vertNum);
		break;
	case SHADOWMAP_SHADER:
		ShadowMapVert << <blockNum, threadNum >> > (cudaVertDatas, cudaFragDatas, cudaDataTruck, vertNum);
		break;
	case PBR_SHADER:
		PBRVert << <blockNum, threadNum >> > (cudaVertDatas, cudaFragDatas, cudaDataTruck, vertNum);
		break;
	case SKYBOX_SHADER:
		SkyBoxVert << <blockNum, threadNum >> > (cudaVertDatas, cudaFragDatas, cudaDataTruck, vertNum);
		break;
	}


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//ͬ��
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//�������Device������Host
	cudaStatus = hipMemcpy(fragDatas->data(), cudaFragDatas, vertNum * sizeof(Varyings), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(cudaVertDatas);
	hipFree(cudaFragDatas);
	hipFree(cudaDataTruck);
	hipFree(cudatmptextures);
	return cudaStatus;
}
/*
* ������ɫ
* �����豸���ڴ�&���к˺���&д������&�ͷ��豸���ڴ�
*/
hipError_t FragKernel(FrameBuffer frameBuffer, std::vector<Varyings>* fragDatas, DataTruck* dataTruck, int shaderID)
{
	hipError_t cudaStatus;

	Varyings* cudaFragDatas = nullptr;
	DataTruck* cudaDataTruck = nullptr;
	Texture* cudatmptextures = nullptr;
	auto tmpDataTruck = *dataTruck;
	
	const int threadNum = 192;
	const int kernelLimit = 1920;
	int trangleNum = fragDatas->size() / 3;
	int blockNum = kernelLimit / threadNum;
	int tnum = 0;


	//fragData
	hipMalloc((void**)&cudaFragDatas, fragDatas->size() * sizeof(Varyings));
	hipMemcpy(cudaFragDatas, fragDatas->data(), fragDatas->size() * sizeof(Varyings), hipMemcpyHostToDevice);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFragDatas fail!");
		goto Error;
	}

	//dataTruck
	cudaStatus = hipMalloc((void**)&cudatmptextures, dataTruck->texNum * sizeof(Texture));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed! cudatmptextures");
		goto Error;
	}
	cudaStatus = hipMemcpy(cudatmptextures, dataTruck->textures, dataTruck->texNum * sizeof(Texture), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemecpy failed! cudatmptextures");
		goto Error;
	}

	tmpDataTruck.textures = cudatmptextures;

	cudaStatus = hipMalloc((void**)&cudaDataTruck, sizeof(DataTruck));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed! cudaDataTruck");
		goto Error;
	}
	cudaStatus = hipMemcpy(cudaDataTruck, &tmpDataTruck, sizeof(DataTruck), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed! cudaDataTruck");
		goto Error;
	}


	while (tnum < trangleNum)
	{
		CaculateTrangle << <blockNum, threadNum >> > (frameBuffer, cudaFragDatas, cudaDataTruck, shaderID, trangleNum, tnum);

		tnum += kernelLimit;

	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching CaculateTrangle!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(cudaFragDatas);
	hipFree(cudatmptextures);
	hipFree(cudaDataTruck);
	return cudaStatus;
}



