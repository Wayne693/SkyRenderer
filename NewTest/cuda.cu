#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include "Model.h"

__global__ void CalVert(std::vector<Attributes>* vertDatas, std::vector<Varyings>* fragDatas, DataTruck* dataTruck, Varyings(*Vert) (Attributes))
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= vertDatas->size())
	{
		return;
	}
	(*fragDatas)[idx] = Vert((*vertDatas)[idx]);
}

hipError_t VertKernel(std::vector<Attributes>* vertDatas, std::vector<Attributes>* fragDatas, DataTruck* dataTruck, Shader* shader)
{
	std::vector<Attributes>* cudaVertDatas = nullptr;
	//Shader* cudaShader = nullptr;
	std::vector<Attributes>* cudaFragDatas = nullptr;
	DataTruck* cudaDataTruck = nullptr;
	std::vector<Texture*>* cudaTextures = nullptr;

	hipError_t cudaStatus;
	const int threadNum = 192;
	int blockNum = vertDatas->size() / threadNum + (cudaVertDatas->size() % threadNum);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//��GPU�����ڴ�
	{
		cudaStatus = hipMalloc((void**)&cudaVertDatas, sizeof(Attributes) * cudaVertDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipMalloc((void**)&cudaShader, sizeof(Shader));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}*/
		cudaStatus = hipMalloc((void**)&cudaFragDatas, sizeof(Varyings) * cudaFragDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&cudaDataTruck, sizeof(DataTruck));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		auto textures = dataTruck->mesh->GetTextures();
		//��cudaTexturesΪ�׵�ַ����TextureList���ڴ�
		cudaStatus = hipMalloc((void**)&cudaTextures, sizeof(Texture*) * textures->size());

		for (int i = 0; i < textures->size(); i++)
		{
			auto currentTexture = (*cudaTextures)[i];
			cudaStatus = hipMalloc((void**)&currentTexture, sizeof(Texture));
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			auto rawData = currentTexture->m_RawBuffer;
			cudaStatus = hipMalloc((void**)&rawData, sizeof((*textures)[i]->m_RawBuffer));
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
		}
		
	}


	//���ݿ���
	{
		cudaStatus = hipMemcpy(cudaVertDatas, vertDatas, sizeof(Attributes) * cudaVertDatas->size(), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		/*cudaStatus = hipMemcpy(cudaShader, shader, sizeof(Shader), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}*/
		cudaStatus = hipMemcpy(cudaDataTruck, dataTruck, sizeof(DataTruck), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		auto textures = dataTruck->mesh->GetTextures();
		cudaStatus = hipMalloc((void**)&cudaTextures, sizeof(Texture*) * textures->size());
		for (int i = 0; i < textures->size(); i++)
		{
			auto currentTexture = (*cudaTextures)[i];
			cudaStatus = hipMalloc((void**)&currentTexture, sizeof(Texture));
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			auto rawData = currentTexture->m_RawBuffer;
			cudaStatus = hipMalloc((void**)&rawData, sizeof((*textures)[i]->m_RawBuffer));
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
		}
	}


	//����Kernel����
	CalVert <<<blockNum, threadNum >>> (cudaVertDatas, cudaShader, cudaDataTruck, );

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//ͬ��
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//�������Device������Host
	cudaStatus = hipMemcpy(fragDatas, cudaFragDatas, sizeof(Varyings) * cudaFragDatas->size(), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(cudaVertDatas);
	//hipFree(cudaShader);
	hipFree(cudaFragDatas);
	hipFree(cudaDataTruck);
}