#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include "Model.h"
#include "Dense"
#include "thrust/extrema.h"

__device__ FrameBuffer* cudaBuffer = nullptr;

//�������Ĳ�ֵ������1-u-v,u,v
__host__ __device__ Eigen::Vector3f barycentric(Eigen::Vector2f A, Eigen::Vector2f B, Eigen::Vector2f C, Eigen::Vector2f P)
{
	Eigen::Vector3f u = Eigen::Vector3f(B.x() - A.x(), C.x() - A.x(), A.x() - P.x()).cross(Eigen::Vector3f(B.y() - A.y(), C.y() - A.y(), A.y() - P.y()));// u v 1
	return Eigen::Vector3f(1.f - (u.x() + u.y()) / u.z(), u.x() / u.z(), u.y() / u.z());
}

__global__ void LambertVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];
	Varyings o;
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = dataTruck->matrixVP * o.positionWS;
	//��normalOSת��normalWS
	Eigen::Matrix3f normalMatrix = dataTruck->matrixM.block(0, 0, 3, 3).transpose();
	Eigen::Matrix3f nn = normalMatrix.inverse();
	o.normalWS = nn * vertex.normalOS;
	//����tangentWS��binormalWS
	o.tangentWS = dataTruck->matrixM.block(0, 0, 3, 3) * vertex.tangentOS.head(3);
	o.binormalWS = o.normalWS.cross(o.tangentWS) * vertex.tangentOS.w();
	//������uv���괦���
	o.uv = TransformTex(vertex.uv, &dataTruck->textures[0]);
	fragDatas[idx] = o;
}

__global__ void PBRVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];
	Varyings o;
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = dataTruck->matrixVP * o.positionWS;
	//��normalOSת��normalWS
	Eigen::Matrix3f normalMatrix = dataTruck->matrixM.block(0, 0, 3, 3).transpose();
	Eigen::Matrix3f nn = normalMatrix.inverse();
	o.normalWS = nn * vertex.normalOS;
	//����tangentWS��binormalWS
	o.tangentWS = dataTruck->matrixM.block(0, 0, 3, 3) * vertex.tangentOS.head(3);
	o.binormalWS = o.normalWS.cross(o.tangentWS) * vertex.tangentOS.w();
	//������uv���괦���
	o.uv = TransformTex(vertex.uv, &dataTruck->textures[0]);
	fragDatas[idx] = o;
}

__global__ void ShadowMapVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];
	Varyings o;
	//��positionOSת��positionWS
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = dataTruck->lightMatrixVP * o.positionWS;

	fragDatas[idx] = o;
}

__global__ void SkyBoxVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];

	auto matrixP = dataTruck->camera.m_ProjectionMtx;
	Eigen::Matrix4f matrixV = Eigen::Matrix4f::Zero();
	matrixV << dataTruck->camera.m_ViewMtx.block(0, 0, 3, 3);
	matrixV(3, 3) = 1;

	Varyings o;

	//��positionOSת��positionWS
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = matrixP * matrixV * o.positionWS;
	fragDatas[idx] = o;
}

__global__ void CaculatePixel(Varyings* fragDatas, DataTruck* dataTruck, Varyings* vertA, Varyings* vertB, Varyings* vertC, int x0, int y0, int w, int h)
{
	const int x = x0 + blockIdx.x * blockDim.x + threadIdx.x;
	const int y = y0 + blockIdx.y * blockDim.y + threadIdx.y;

	auto a = ComputeScreenPos(cudaBuffer, vertA->positionCS);
	auto b = ComputeScreenPos(cudaBuffer, vertB->positionCS);
	auto c = ComputeScreenPos(cudaBuffer, vertC->positionCS);

	Eigen::Vector3f u = barycentric(Eigen::Vector2f(a.x(), a.y()), Eigen::Vector2f(b.x(), b.y()), Eigen::Vector2f(c.x(), c.y()), Eigen::Vector2f(x, y));
	if (u.x() >= 0 && u.y() >= 0 && u.z() >= 0)
	{
		float depth = u.x() * a.z() + u.y() * b.z() + u.z() * c.z();

		if (depth > GetZ(&dataTruck->shadowMap, x, y))
		{
			return;
		}

		float alpha = u.x() / vertA->positionCS.w();
		float beta = u.y() / vertB->positionCS.w();
		float gamma = u.z() / vertC->positionCS.w();
		float zn = 1 / (alpha + beta + gamma);

		//��ֵ
		Varyings tmpdata;
		tmpdata.positionWS = zn * (alpha * vertA->positionWS + beta * vertB->positionWS + gamma * vertC->positionWS);
		tmpdata.positionCS = zn * (alpha * vertA->positionCS + beta * vertB->positionCS + gamma * vertC->positionCS);
		tmpdata.normalWS = zn * (alpha * vertA->normalWS + beta * vertB->normalWS + gamma * vertC->normalWS);
		tmpdata.tangentWS = zn * (alpha * vertA->tangentWS + beta * vertB->tangentWS + gamma * vertC->tangentWS);
		tmpdata.binormalWS = zn * (alpha * vertA->binormalWS + beta * vertB->binormalWS + gamma * vertC->binormalWS);
		tmpdata.uv = zn * (alpha * vertA->uv + beta * vertB->uv + gamma * vertC->uv);

		/*****************************************************LambertFrag*****************************************************/

		Varyings i = tmpdata;

		auto mainLight = dataTruck->mainLight;
		Eigen::Vector3f lightDirWS = -1 * mainLight.direction;
		lightDirWS.normalize();

		//����TBN
		Eigen::Matrix3f tbnMatrix;
		tbnMatrix << i.tangentWS.x(), i.binormalWS.x(), i.normalWS.x(),
			i.tangentWS.y(), i.binormalWS.y(), i.normalWS.y(),
			i.tangentWS.z(), i.binormalWS.z(), i.normalWS.z();
		//��÷��������з�������
		Eigen::Vector3f bumpTS = UnpackNormal(&dataTruck->textures[1], i.uv);
		Eigen::Vector3f bumpWS = (tbnMatrix * bumpTS).normalized();

		//diffuse
		float NdotL = bumpWS.dot(lightDirWS);
		Eigen::Vector4f diffuse = mainLight.intensity * thrust::max(NdotL, 0.f) * Vec4Mul(mainLight.color, Tex2D(&dataTruck->textures[0], i.uv));

		float shadow = 0.f;
		Eigen::Vector4f positionLSS = ComputeScreenPos(cudaBuffer, dataTruck->lightMatrixVP * i.positionWS);
		float bias = thrust::max(0.05 * (1 - bumpWS.dot(lightDirWS)), 0.01);
		//PCF
		for (int i = -1; i <= 1; i++)
		{
			for (int j = -1; j <= 1; j++)
			{
				shadow += (positionLSS.z() > GetZ(&dataTruck->shadowMap, positionLSS.x() + i, positionLSS.y() + j) + bias);
			}
		}
		shadow = thrust::min(0.7f, shadow / 9);

		Eigen::Vector4f finalColor = (1 - shadow) * diffuse;

		/*******************************************************д������************************************************************/
		DrawPoint(cudaBuffer, x, y, finalColor);
		SetZ(cudaBuffer, x, y, depth);
	}
}

__global__ void CaculateTrangle(Varyings* fragDatas, DataTruck* dataTruck, int* trangleNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= *trangleNum)
	{
		return;
	}

	auto vertA = fragDatas[idx * 3];
	auto vertB = fragDatas[idx * 3 + 1];
	auto vertC = fragDatas[idx * 3 + 2];

	auto a = ComputeScreenPos(cudaBuffer, vertA.positionCS);
	auto b = ComputeScreenPos(cudaBuffer, vertB.positionCS);
	auto c = ComputeScreenPos(cudaBuffer, vertC.positionCS);
	// caculate AABB box
	int minx = thrust::max(0, thrust::min((int)cudaBuffer->m_Height, (int)thrust::min(a.x(), thrust::min(b.x(), c.x()))));
	int miny = thrust::max(0, thrust::min((int)cudaBuffer->m_Width, (int)thrust::min(a.y(), thrust::min(b.y(), c.y()))));
	int maxx = thrust::min((int)cudaBuffer->m_Width, thrust::max(0, (int)thrust::max(a.x(), thrust::max(b.x(), c.x()))));
	int maxy = thrust::min((int)cudaBuffer->m_Height, thrust::max(0, (int)thrust::max(a.y(), thrust::max(b.y(), c.y()))));
	//AABB��Χ�еĿ��
	int h = maxy - miny + 1;
	int w = maxx - minx + 1;

	const int threadNum = 32;

	dim3 blockNum(w / threadNum + (w % threadNum != 0), h / threadNum + (h % threadNum));
	dim3 blockSize(threadNum, threadNum);

	CaculatePixel <<<blockNum, blockSize >>> (fragDatas, dataTruck, &vertA, &vertB, &vertC, minx, miny, w, h);
	
	
}

hipError_t VertKernel(std::vector<Attributes>* vertDatas, std::vector<Varyings>* fragDatas, DataTruck* dataTruck, int shaderID)
{
	hipError_t cudaStatus;

	Attributes* cudaVertDatas = nullptr;
	Varyings* cudaFragDatas = nullptr;
	DataTruck* cudaDataTruck = nullptr;
	int* cudaVertNum = nullptr;
	Texture* cudatmptextures = nullptr;

	const int threadNum = 192;
	int vertNum = vertDatas->size();
	int blockNum = vertDatas->size() / threadNum + (vertNum % threadNum != 0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//�������ݵ�GPU�ڴ�
	{
		//��������
		cudaStatus = hipMalloc((void**)&cudaVertDatas, sizeof(Attributes) * vertDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaVertDatas");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudaVertDatas, vertDatas->data(), vertNum * sizeof(Attributes), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed! cudaVertDatas");
			goto Error;
		}

		//������ɫ������
		cudaStatus = hipMalloc((void**)&cudaFragDatas, sizeof(Varyings) * fragDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaFragDatas");
			goto Error;
		}
		//dataTruck
		/*
		* �ṹ���к�ָ����������ڴ淽���ο�����
		* https://devforum.nvidia.cn/forum.php?mod=viewthread&tid=6820&extra=&page=1
		*/
		
		cudaStatus = hipMalloc((void**)&cudatmptextures, dataTruck->texNum * sizeof(Texture));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudatmptextures");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudatmptextures, dataTruck->textures, dataTruck->texNum * sizeof(Texture), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "cudaMemecpy failed! cudatmptextures");
			goto Error;
		}
		
		auto tmpDataTruck = *dataTruck;
		tmpDataTruck.textures = cudatmptextures;

		cudaStatus = hipMalloc((void**)&cudaDataTruck, sizeof(DataTruck));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaDataTruck");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudaDataTruck, &tmpDataTruck, sizeof(DataTruck), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed! cudaDataTruck");
			goto Error;
		}

		//vertNum
		cudaStatus = hipMalloc((void**)&cudaVertNum, sizeof(int));
		cudaStatus = hipMemcpy(cudaVertNum, &vertNum, sizeof(int), hipMemcpyHostToDevice);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			printf("cudaFailed vertNum : %s", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}

	//����Kernel����
	switch (shaderID)
	{
	case NONE:
		break;
	case LAMBERT_SHADER:
		LambertVert <<<blockNum, threadNum >>> (cudaVertDatas, cudaFragDatas, cudaDataTruck, cudaVertNum);
		break;
	case SHADOWMAP_SHADER:
		ShadowMapVert <<<blockNum, threadNum >>> (cudaVertDatas, cudaFragDatas, cudaDataTruck, cudaVertNum);
		break;
	case PBR_SHADER:
		PBRVert <<<blockNum, threadNum >>> (cudaVertDatas, cudaFragDatas, cudaDataTruck, cudaVertNum);
		break;
	case SKYBOX_SHADER:
		SkyBoxVert << <blockNum, threadNum >> > (cudaVertDatas, cudaFragDatas, cudaDataTruck, cudaVertNum);
		break;
	}
	

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//ͬ��
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//�������Device������Host
	cudaStatus = hipMemcpy(fragDatas->data(), cudaFragDatas, vertNum * sizeof(Varyings), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(cudaVertDatas);
	hipFree(cudaFragDatas);
	hipFree(cudaDataTruck);
	hipFree(cudatmptextures);
	hipFree(cudaVertNum);
	return cudaStatus;
}

hipError_t FragKernel(std::vector<Varyings>* fragDatas, DataTruck* dataTruck, int shaderID)
{
	hipError_t cudaStatus;

	Varyings* cudaFragDatas = nullptr;
	DataTruck* cudaDataTruck = nullptr;
	Texture* cudatmptextures = nullptr;
	int* cudaTrangleNum = nullptr;
	auto tmpDataTruck = *dataTruck;

	const int threadNum = 192;
	int trangleNum = fragDatas->size() / 3;
	int blockNum = trangleNum / threadNum + (trangleNum % threadNum != 0);
	
	//fragData
	hipMalloc((void**)&cudaFragDatas, fragDatas->size() * sizeof(Varyings));
	hipMemcpy((void**)&cudaFragDatas, fragDatas->data(), fragDatas->size() * sizeof(Varyings), hipMemcpyDeviceToHost);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFragDatas fail!");
		goto Error;
	}

	//dataTruck
	cudaStatus = hipMalloc((void**)&cudatmptextures, dataTruck->texNum * sizeof(Texture));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed! cudatmptextures");
		goto Error;
	}
	cudaStatus = hipMemcpy(cudatmptextures, dataTruck->textures, dataTruck->texNum * sizeof(Texture), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemecpy failed! cudatmptextures");
		goto Error;
	}

	tmpDataTruck.textures = cudatmptextures;

	cudaStatus = hipMalloc((void**)&cudaDataTruck, sizeof(DataTruck));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed! cudaDataTruck");
		goto Error;
	}
	cudaStatus = hipMemcpy(cudaDataTruck, &tmpDataTruck, sizeof(DataTruck), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed! cudaDataTruck");
		goto Error;
	}

	//trangleNum
	hipMalloc((void**)&cudaTrangleNum, sizeof(int));
	hipMemcpy(cudaTrangleNum, &trangleNum, sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaTrangleNum fail!");
		goto Error;
	}

	CaculateTrangle <<<blockNum, threadNum >>> (cudaFragDatas,cudaDataTruck,cudaTrangleNum);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(cudaFragDatas);
	hipFree(cudatmptextures);
	hipFree(cudaDataTruck);
	hipFree(cudaTrangleNum);
	return cudaStatus;
}




hipError_t LoadFrameBuffer(FrameBuffer* frameBuffer)
{
	hipError_t cudaStatus;

	hipMalloc((void**)&cudaBuffer, sizeof(FrameBuffer));
	hipMemcpy(cudaBuffer, frameBuffer, sizeof(FrameBuffer), hipMemcpyHostToDevice);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaDisplay Failed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	return cudaStatus;
}

void CudaFreeFrameBuffer()
{
	hipFree(cudaBuffer);
}
