#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include "Model.h"
#include "Dense"

__global__ void CalVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];
	Varyings o;
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = dataTruck->matrixVP * o.positionWS;
	//��normalOSת��normalWS
	Eigen::Matrix3f normalMatrix = dataTruck->matrixM.block(0, 0, 3, 3).transpose();
	Eigen::Matrix3f nn = normalMatrix.inverse();
	o.normalWS = nn * vertex.normalOS;
	//����tangentWS��binormalWS
	o.tangentWS = dataTruck->matrixM.block(0, 0, 3, 3) * vertex.tangentOS.head(3);
	o.binormalWS = o.normalWS.cross(o.tangentWS) * vertex.tangentOS.w();
	//������uv���괦���
	float x = vertex.uv.x();
	float y = vertex.uv.y(); 
	o.uv = Eigen::Vector2f(x, y);

	fragDatas[idx] = o;
}


hipError_t VertKernel(std::vector<Attributes>* vertDatas, std::vector<Varyings>* fragDatas, DataTruck* dataTruck, Shader* shader)
{
	Attributes* cudaVertDatas = nullptr;
	Varyings* cudaFragDatas = nullptr;
	DataTruck* cudaDataTruck = nullptr;
	int* cudaVertNum = nullptr;
	//Texture** cudaTextures = nullptr;

	hipError_t cudaStatus;
	const int threadNum = 192;
	int vertNum = vertDatas->size();
	int blockNum = vertDatas->size() / threadNum + (vertNum % threadNum != 0);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//��GPU�����ڴ�
	{
		cudaStatus = hipMalloc((void**)&cudaVertDatas, sizeof(Attributes) * vertDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&cudaFragDatas, sizeof(Varyings) * fragDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&cudaDataTruck, sizeof(DataTruck));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		hipMalloc((void**)&cudaVertNum, sizeof(int));
		hipMemcpy(cudaVertNum, &vertNum, sizeof(int), hipMemcpyHostToDevice);
		//auto textures = dataTruck->mesh->GetTextures();
		////��cudaTexturesΪ�׵�ַ����TextureList���ڴ�
		//cudaStatus = hipMalloc((void**)&cudaTextures, sizeof(Texture*) * textures->size());
		////printf("%d\n", sizeof(*cudaTextures));
		//for (int i = 0; i < textures->size(); i++)
		//{
		//	//Texture* currentTextureAdd = ;
		//	cudaStatus = hipMalloc((void**)cudaTextures[i], sizeof(Texture));
		//	hipMemcpy(cudaTextures[i], (*textures)[i], sizeof(Texture), hipMemcpyHostToDevice);

		//	auto rawData = cudaTextures[i]->m_RawBuffer;
		//	cudaStatus = hipMalloc((void**)&cudaTextures[i], sizeof((*textures)[i]->m_RawBuffer));
		//	hipMemcpy(rawData, (*textures)[i]->m_RawBuffer, sizeof((*textures)[i]->m_RawBuffer), hipMemcpyHostToDevice);
		//}

	}


	//���ݿ���
	{
		cudaStatus = hipMemcpy(cudaVertDatas, vertDatas->data(), vertNum * sizeof(Attributes), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudaDataTruck, dataTruck, sizeof(DataTruck), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	}

	//����Kernel����
	CalVert <<<blockNum, threadNum >>> (cudaVertDatas, cudaFragDatas, cudaDataTruck,cudaVertNum);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//ͬ��
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//�������Device������Host
	cudaStatus = hipMemcpy(fragDatas->data(), cudaFragDatas, vertNum * sizeof(Varyings), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(cudaVertDatas);
	hipFree(cudaFragDatas);
	hipFree(cudaDataTruck);

	return cudaStatus;
}

