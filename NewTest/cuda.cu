#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include "Model.h"
#include "Dense"

uint32_t* cudaTexData;
int* cudaTexOffset;
uint32_t* cudaBufData;
int* cudaBufOffset;

__global__ void LambertVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];
	Varyings o;
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = dataTruck->matrixVP * o.positionWS;
	//��normalOSת��normalWS
	Eigen::Matrix3f normalMatrix = dataTruck->matrixM.block(0, 0, 3, 3).transpose();
	Eigen::Matrix3f nn = normalMatrix.inverse();
	o.normalWS = nn * vertex.normalOS;
	//����tangentWS��binormalWS
	o.tangentWS = dataTruck->matrixM.block(0, 0, 3, 3) * vertex.tangentOS.head(3);
	o.binormalWS = o.normalWS.cross(o.tangentWS) * vertex.tangentOS.w();
	//������uv���괦���
	o.uv = TransformTex(vertex.uv, &dataTruck->textures[0]);
	fragDatas[idx] = o;
}

__global__ void PBRVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];
	Varyings o;
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = dataTruck->matrixVP * o.positionWS;
	//��normalOSת��normalWS
	Eigen::Matrix3f normalMatrix = dataTruck->matrixM.block(0, 0, 3, 3).transpose();
	Eigen::Matrix3f nn = normalMatrix.inverse();
	o.normalWS = nn * vertex.normalOS;
	//����tangentWS��binormalWS
	o.tangentWS = dataTruck->matrixM.block(0, 0, 3, 3) * vertex.tangentOS.head(3);
	o.binormalWS = o.normalWS.cross(o.tangentWS) * vertex.tangentOS.w();
	//������uv���괦���
	o.uv = TransformTex(vertex.uv, &dataTruck->textures[0]);
	fragDatas[idx] = o;
}

__global__ void ShadowMapVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];
	Varyings o;
	//��positionOSת��positionWS
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = dataTruck->lightMatrixVP * o.positionWS;

	fragDatas[idx] = o;
}

__global__ void SkyBoxVert(Attributes* vertDatas, Varyings* fragDatas, DataTruck* dataTruck, int* vertNum)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx >= *vertNum)
	{
		return;
	}
	Attributes vertex = vertDatas[idx];

	auto matrixP = dataTruck->camera.m_ProjectionMtx;
	Eigen::Matrix4f matrixV = Eigen::Matrix4f::Zero();
	matrixV << dataTruck->camera.m_ViewMtx.block(0, 0, 3, 3);
	matrixV(3, 3) = 1;

	Varyings o;

	//��positionOSת��positionWS
	o.positionWS = dataTruck->matrixM * vertex.positionOS;
	//��positionWSת��positionCS
	o.positionCS = matrixP * matrixV * o.positionWS;
	fragDatas[idx] = o;
}

//������������ú˺������ͷ��ڴ�
hipError_t VertKernel(std::vector<Attributes>* vertDatas, std::vector<Varyings>* fragDatas, DataTruck* dataTruck, int shaderID)
{
	hipError_t cudaStatus;


	Attributes* cudaVertDatas = nullptr;
	Varyings* cudaFragDatas = nullptr;
	DataTruck* cudaDataTruck = nullptr;
	int* cudaVertNum = nullptr;
	Texture* cudatmptextures = nullptr;

	const int threadNum = 192;
	int vertNum = vertDatas->size();
	int blockNum = vertDatas->size() / threadNum + (vertNum % threadNum != 0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//�������ݵ�GPU�ڴ�
	{
		//��������
		cudaStatus = hipMalloc((void**)&cudaVertDatas, sizeof(Attributes) * vertDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaVertDatas");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudaVertDatas, vertDatas->data(), vertNum * sizeof(Attributes), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed! cudaVertDatas");
			goto Error;
		}

		//������ɫ������
		cudaStatus = hipMalloc((void**)&cudaFragDatas, sizeof(Varyings) * fragDatas->size());
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaFragDatas");
			goto Error;
		}
		//dataTruck
		/*
		* �ṹ���к�ָ����������ڴ淽���ο�����
		* https://devforum.nvidia.cn/forum.php?mod=viewthread&tid=6820&extra=&page=1
		*/
		
		cudaStatus = hipMalloc((void**)&cudatmptextures, dataTruck->texNum * sizeof(Texture));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudatmptextures");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudatmptextures, dataTruck->textures, dataTruck->texNum * sizeof(Texture), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "cudaMemecpy failed! cudatmptextures");
			goto Error;
		}
		
		auto tmpDataTruck = *dataTruck;
		tmpDataTruck.textures = cudatmptextures;

		cudaStatus = hipMalloc((void**)&cudaDataTruck, sizeof(DataTruck));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed! cudaDataTruck");
			goto Error;
		}
		cudaStatus = hipMemcpy(cudaDataTruck, &tmpDataTruck, sizeof(DataTruck), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed! cudaDataTruck");
			goto Error;
		}

		//vertNum
		cudaStatus = hipMalloc((void**)&cudaVertNum, sizeof(int));
		cudaStatus = hipMemcpy(cudaVertNum, &vertNum, sizeof(int), hipMemcpyHostToDevice);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			printf("cudaFailed vertNum : %s", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}

	//����Kernel����
	switch (shaderID)
	{
	case NONE:
		break;
	case LAMBERT_SHADER:
		LambertVert <<<blockNum, threadNum >>> (cudaVertDatas, cudaFragDatas, cudaDataTruck, cudaVertNum);
		break;
	case SHADOWMAP_SHADER:
		ShadowMapVert <<<blockNum, threadNum >>> (cudaVertDatas, cudaFragDatas, cudaDataTruck, cudaVertNum);
		break;
	case PBR_SHADER:
		PBRVert <<<blockNum, threadNum >>> (cudaVertDatas, cudaFragDatas, cudaDataTruck, cudaVertNum);
		break;
	case SKYBOX_SHADER:
		SkyBoxVert << <blockNum, threadNum >> > (cudaVertDatas, cudaFragDatas, cudaDataTruck, cudaVertNum);
		break;
	}
	

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//ͬ��
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//�������Device������Host
	cudaStatus = hipMemcpy(fragDatas->data(), cudaFragDatas, vertNum * sizeof(Varyings), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(cudaVertDatas);
	hipFree(cudaFragDatas);
	hipFree(cudaDataTruck);
	hipFree(cudatmptextures);
	hipFree(cudaVertNum);
	return cudaStatus;
}

//������������(RenderLoopǰ����)
hipError_t LoadTextureData(std::vector<uint32_t>* rawData, std::vector<int>* offset)
{
	hipError_t cudaStatus;

	hipMalloc((void**)&cudaTexData, rawData->size() * sizeof(uint32_t));
	hipMalloc((void**)&cudaTexOffset, offset->size() * sizeof(int));
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipMemcpy(cudaTexData, rawData->data(), rawData->size() * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(cudaTexOffset, offset->data(), offset->size() * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	return cudaStatus;
}

//����FrameBuffer����(ÿ֡����)
hipError_t LoadBufferData(std::vector<uint32_t>* rawData, std::vector<int>* offset)
{
	hipError_t cudaStatus;

	hipMalloc((void**)&cudaBufData, rawData->size() * sizeof(uint32_t));
	hipMalloc((void**)&cudaBufOffset, offset->size() * sizeof(int));
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipMemcpy(cudaBufData, rawData->data(), rawData->size() * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(cudaBufOffset, offset->data(), offset->size() * sizeof(int), hipMemcpyHostToDevice);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	return cudaStatus;
}

void CudaFreeBufferData()
{
	hipFree(cudaBufData);
	hipFree(cudaBufOffset);
}