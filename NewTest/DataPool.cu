#include "hip/hip_runtime.h"
#include"DataPool.h"

/*
*����������
* CPU��ʹ��
*/
//��������
 int textureNum;
 std::vector<int> textureOffset;
 std::vector<uint32_t> textureRawData;

//FrameBuffer����
 int bufferNum;
 std::vector<int> bufferOffset;
 std::vector<uint32_t> bufferData;

 /*
 * �Դ�������
 * GPU��ʹ��
 */
 __device__ uint32_t* cudaTexData;
 __device__ int* cudaTexOffset;
 __device__ uint32_t* cudaBufData;
 __device__ int* cudaBufOffset;
 //������ת���ͷ�device�ڴ�ı���
 uint32_t* cudaBufDataHost = nullptr;
 int* cudaBufOffsetHost = nullptr;

//low-level API
__host__ __device__  uint32_t GetData(uint32_t* rawData, int* offset, int id, int pos)
{
	return rawData[offset[id] + pos];
}

__host__ __device__  void SetData(uint32_t* rawData, int* offset, int id, int pos, uint32_t val)
{
	rawData[offset[id] + pos] = val;
}

 int AddData(std::vector<uint32_t>& dstRawData, std::vector<int>& offset, uint32_t* srcRawData, int size)
{
	if (size <= 0)
	{
		return -1;
	}
	int datasize = dstRawData.size();
	dstRawData.resize(datasize + size);
	//��Դ����Ϊ��ָ����ֻ����ռ䲻��������
	if (srcRawData != nullptr)
	{
		memcpy(dstRawData.data() + datasize, srcRawData, size * sizeof(uint32_t));
	}
	offset.push_back(datasize);
	return 0;
}

 void initData(uint32_t* dstData, uint32_t val, int size)
{
	std::fill_n(dstData, size, val);
}

 __device__ float fatomicMin(uint32_t* addr, float value)
 {
	 uint32_t old = *addr, assumed;
	 //printf("*up* addr = %lf value = %lf\n", __int_as_float(*addr), value);
	 //if (old <= value) return old;
	 do
	 {
		 assumed = old;
		 old = atomicCAS(addr, assumed, __float_as_int(fminf(value, __int_as_float(assumed))));
		 //printf("%")
	 } while (old != assumed);
	 //printf("*down* addr = %lf old = %lf\n", __int_as_float(*addr), __int_as_float(old));
	 return __int_as_float(*addr);
 }

//high-level API
//�����������ݺ���
 int AddTextureData(uint32_t* rawData, int size)
{
	if (AddData(textureRawData, textureOffset, rawData, size) == 0)
	{
		return textureNum++;
	}
	printf("add texture fail\n");
	return -1;
}

 std::vector<uint32_t>* RawData()
{
	return &textureRawData;
}

 std::vector<int>* Offset()
{
	return &textureOffset;
}

uint32_t GetRawData(int id, int pos)
{
	return GetData(textureRawData.data(), textureOffset.data(), id, pos);
}

__device__ uint32_t CudaGetRawData(int id, int pos)
{
	return GetData(cudaTexData, cudaTexOffset, id, pos);
}

void SetRawData(int id, int pos, uint32_t color)
{
	SetData(textureRawData.data(), textureOffset.data(), id, pos, color);
}

__device__ void CudaSetRawData(int id, int pos, uint32_t color)
{
	SetData(cudaTexData, cudaTexOffset, id, pos, color);
}

//����FrameBuffer���ݺ���
std::vector<uint32_t>* BufferData()
{
	return &bufferData;
}

std::vector<int>* BufferOffset()
{
	return &bufferOffset;
}

int AddBufferData(uint32_t* rawData, int size)
{
	if (AddData(bufferData, bufferOffset, rawData, size) == 0)
	{
		return bufferNum++;
	}
	printf("add framebuffer fail\n");
	return -1;
}

uint32_t GetBufferData(int id, int pos)
{
	return GetData(bufferData.data(), bufferOffset.data(), id, pos);
}

__device__ uint32_t CudaGetBufferData(int id, int pos)
{
	return GetData(cudaBufData, cudaBufOffset, id, pos);
}

__device__ float MinZAutomic(int id, int pos, float depth)
{
	return fatomicMin(cudaBufData + cudaBufOffset[id] + pos, depth);
}

uint32_t* GetBuffer(int id)
{
	return bufferData.data() + bufferOffset[id];
}

void SetBufferData(int id, int pos, uint32_t color)
{
	SetData(bufferData.data(), bufferOffset.data(), id, pos, color);
}

__device__ void CudaSetBufferData(int id, int pos, uint32_t color)
{
	SetData(cudaBufData, cudaBufOffset, id, pos, color);
}

void ClearBuffer(int id, int size, uint32_t color) 
{
	initData(bufferData.data() + bufferOffset[id], color, size);
}

void ClearBuffer(int id, int size, float color)
{
	initData(bufferData.data() + bufferOffset[id], *(uint32_t*)&color, size);
}


hipError_t LoadTextureData(std::vector<uint32_t>* rawData, std::vector<int>* offset)
{
	hipError_t cudaStatus;

	//������ת����ʱ����
	uint32_t* cudaTexDataTmp = nullptr;
	int* cudaTexOffsetTmp = nullptr;

	//����ʱ������device�˷����ڴ�
	hipMalloc((void**)&cudaTexDataTmp, rawData->size() * sizeof(uint32_t));
	hipMalloc((void**)&cudaTexOffsetTmp, offset->size() * sizeof(int));
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	//����ʱ������device�˿�������
	hipMemcpy(cudaTexDataTmp, rawData->data(), rawData->size() * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(cudaTexOffsetTmp, offset->data(), offset->size() * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//����ʱ���������ַ��ֵ��device����
	hipMemcpyToSymbol(HIP_SYMBOL(cudaTexData), &cudaTexDataTmp, sizeof(uint32_t*));
	hipMemcpyToSymbol(HIP_SYMBOL(cudaTexOffset), &cudaTexOffsetTmp, sizeof(int*));
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}
Error:
	return cudaStatus;
}


hipError_t LoadBufferData(std::vector<uint32_t>* rawData, std::vector<int>* offset)
{
	hipError_t cudaStatus;

	//����ʱ������device�˷����ڴ�
	hipMalloc((void**)&cudaBufDataHost, rawData->size() * sizeof(uint32_t));
	hipMalloc((void**)&cudaBufOffsetHost, offset->size() * sizeof(int));
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaBufData Failed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//����ʱ������device�˿�������
	hipMemcpy(cudaBufDataHost, rawData->data(), rawData->size() * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(cudaBufOffsetHost, offset->data(), offset->size() * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaBufOffsetFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//����ʱ���������ַ��ֵ��device����
	hipMemcpyToSymbol(HIP_SYMBOL(cudaBufData), &cudaBufDataHost, sizeof(uint32_t*));
	hipMemcpyToSymbol(HIP_SYMBOL(cudaBufOffset), &cudaBufOffsetHost, sizeof(int*));
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaFailed : %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	return cudaStatus;
}

hipError_t LoadBufferDeviceToHost()
{
	hipMemcpy(bufferData.data(), cudaBufDataHost, bufferData.size() * sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipMemcpy(bufferOffset.data(), cudaBufOffsetHost, bufferOffset.size() * sizeof(int), hipMemcpyDeviceToHost);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("LoadBufferDataDeviceToHostFailed : %s", hipGetErrorString(cudaStatus));
	}
	return cudaStatus;
}	

void CudaFreeBufferData()
{
	hipFree(cudaBufDataHost);
	hipFree(cudaBufOffsetHost);
}